#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudastart.cuh"


// kernel function
// each thread calculates one item in matrix 
__global__ void sumMatrix2D(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    int ix = threadIdx.x+blockDim.x*blockIdx.x;
    int iy = threadIdx.y+blockDim.y*blockIdx.y;
    int idx = ix+iy*ny;
    if (ix<nx && iy<ny)
    {
        MatC[idx] = MatA[idx]+MatB[idx];
    }
}

extern "C" {
    void sumMatrix2DGPU(float* C_dev, float* A_dev, float* B_dev, int nx, int ny) {
        double gpuStart = cpuSecond();
        // initDevice(0); // This should be caller's concern
        // 2-d bolck ，32×32
        dim3 block(32, 32);
        // 2-d grid，128×128
        dim3 grid((nx-1)/block.x+1, (ny-1)/block.y+1);


        //将核函数放在线程网格中执行
        sumMatrix2D<<<grid,block>>>(A_dev, B_dev, C_dev, nx, ny);
        CHECK(hipDeviceSynchronize());
        // hipDeviceReset();
        double gpuTime = cpuSecond() - gpuStart;
        printf("GPU matrix addition in C costs: %f sec\n", gpuTime);
    }
}